#include "hip/hip_runtime.h"
// standard libraries
#include <chrono>
#include <execution>
#include <algorithm>
#include <filesystem>

// custom headers
#include "monte_carlo.cuh"
#include "kernels.cuh"
#include "h5_helper.h"
#include "barkeep.h"

#include "definitions.h"

// boost headers
#include <boost/log/trivial.hpp>

// CUDA libraries
#ifdef __HIPCC__
#include "device_helper.cuh"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#endif


#define BLOCKS  256

namespace bl = boost::log;

namespace sim
{

monte_carlo::monte_carlo(bool gpu_disabled, int32_t device_id)
{
    this->gpu_disabled = true;
#ifdef __HIPCC__
    this->gpu_disabled  = gpu_disabled;
    if(gpu_disabled == false){
        if(sim::check_CUDA() == false){
            std::cout << WARN_MSG << "No GPU Device found! switching to CPU mode." << std::endl;
            this->gpu_disabled = gpu_disabled = true;
        }
    }
    if(gpu_disabled == false){
        uint32_t device_count = sim::get_device_count();
        if (device_id >= device_count){
            std::cout << ERR_MSG << "Device ID " << device_id << " is not available! Number of available GPU(s) is " << device_count << " ,switching to CPU mode!" << std::endl;
            this->gpu_disabled = gpu_disabled = true;
        } else {
            BOOST_LOG_TRIVIAL(info) << "Number of available GPU(s): " << device_count; 
            hipSetDevice(device_id);
        }
    }
#endif  
}

monte_carlo::~monte_carlo()
{
}

void monte_carlo::allocate_memory()
{
    size_t trj_dim_size  = param.enRecordTrajectory ? param.n_timepoints * (param.n_dummy_scan + 1) : 1;
    XYZ0.resize(param.n_spins * 3);     // memory layout(row-major): [n_spins x 3]
    XYZ0_scaled.resize(XYZ0.size());       // memory layout(row-major): [n_spins x 3]
    XYZ1.resize(trj_dim_size * param.n_scales * XYZ0.size());     // memory layout(row-major): [n_scale x n_spins x timepoints x 3] or [n_scale x n_spins x 1 x 3]
    M0.resize(param.n_spins * 3);       // memory layout(row-major): [n_spins x 3]
    M1.resize(param.n_scales * param_hvec.TE_us.size() * M0.size());    // memory layout(row-major): [n_scale x n_spins x n_TE x 3]
    T.resize(M1.size()/3);                 // memory layout(row-major): [n_scale x n_spins x n_TE x 1]
}

size_t monte_carlo::get_total_memory() const
{
    size_t total_memory = 0;
    total_memory += XYZ0_scaled.size()  * sizeof(float);
    total_memory += XYZ1.size()         * sizeof(float);
    total_memory += M0.size()           * sizeof(float);
    total_memory += M1.size()           * sizeof(float);
    total_memory += T.size()            * sizeof(uint8_t);
    total_memory += mask.size()         * sizeof(uint8_t);
    total_memory += fieldmap.size()     * sizeof(float);
    return total_memory >> 20; // convert to MB
}

bool monte_carlo::read_phantom(std::string filename)
{
    fov.resize(3);
    if(h5_helper::read(filename, "fieldmap", true, fieldmap) == false)
        fieldmap.clear();
    if(h5_helper::read(filename, "mask", true, mask) == false)
        return false;   
    if(h5_helper::read(filename, "fov", false, fov) == false)
        return false;   

    std::vector<size_t> phantom_size;  
    if(h5_helper::size(filename, "mask", phantom_size) == false)
        return false;
    std::copy(phantom_size.begin(), phantom_size.end(), param.phantom_size);

    uint32_t n_substrate = *std::max_element(std::execution::par, mask.begin(), mask.end()) + 1;
    if (n_substrate > param.n_substrate)
    {
        BOOST_LOG_TRIVIAL(error) << "The number of substrate types in the mask does not match the number of substrate types in the config file: " << n_substrate << " vs " << param.n_substrate;
        return false;
    }

    BOOST_LOG_TRIVIAL(info) << "Size = " << phantom_size[0] << " x " << phantom_size[1] << " x " << phantom_size[2] << std::endl;
    BOOST_LOG_TRIVIAL(info) << "FoV = " << fov[0]*1e6 << " x " << fov[1]*1e6 << " x " << fov[2]*1e6 << " um^3" << std::endl;
    return true;
}

bool monte_carlo::initialize_position(std::string filename, size_t seed)
{
    BOOST_LOG_TRIVIAL(info) << "Initializing positions...";
    if (filename.empty() == false){
        BOOST_LOG_TRIVIAL(info) << "Reading initial positions from file: " << filename;
        if(h5_helper::read(filename, "XYZ", false, XYZ0) == false)
            return false; 
        // check values from file are in FoV
        uint32_t ind=0;
        if (std::any_of(std::execution::par, XYZ0.begin(), XYZ0.end(), [this, &ind](float x){return x < 0 || x > fov[ind++%3];})) {
            BOOST_LOG_TRIVIAL(error) << "Initial positions are outside the FoV.";
            return false;
        }
        return true;
    }
    // if no filename is provided, generate random positions
    BOOST_LOG_TRIVIAL(info) << "Generating random positions within 98% of the FoV with seed = " << seed;
    std::mt19937 gen(seed);
    std::uniform_real_distribution<float> dist_initial_x(0.01*fov[0], 0.99*fov[0]);
    std::uniform_real_distribution<float> dist_initial_y(0.01*fov[1], 0.99*fov[1]);
    std::uniform_real_distribution<float> dist_initial_z(0.01*fov[2], 0.99*fov[2]);

    for (size_t i = 0; i < XYZ0.size() / 3; i++){
        XYZ0[3*i+0] = dist_initial_x(gen);
        XYZ0[3*i+1] = dist_initial_y(gen);
        XYZ0[3*i+2] = dist_initial_z(gen);
    }    
    return true;
}

bool monte_carlo::initialize_magnetization(std::string filename)
{
    BOOST_LOG_TRIVIAL(info) << "Initializing magnetization...";
    if (filename.empty() == false)
        if(h5_helper::read(filename, "M", false, M0) == false)
            return false; 

    BOOST_LOG_TRIVIAL(info) << "Generating M0(0, 0, 1)..." << std::endl;
    uint32_t index = 0;
    std::generate(M0.begin(), M0.end(), [&index](){return (index++ % 3 == 2) ? 1.f : 0.f;});
    return true;
}

void monte_carlo::save(std::string filename)
{
#ifdef __HIPCC__
    if(gpu_disabled == false){
        thrust::copy(d_M1.begin(), d_M1.end(), M1.begin());
        thrust::copy(d_XYZ1.begin(), d_XYZ1.end(), XYZ1.begin());
        thrust::copy(d_T.begin(), d_T.end(), T.begin());
    }
#endif

    std::vector<size_t> dims = {param.n_scales, param.n_spins, param_hvec.TE_us.size(), 3};
    h5_helper::write(filename, "M", dims, M1);

    dims[2] = param.enRecordTrajectory ? param.n_timepoints * (param.n_dummy_scan + 1) : 1;
    h5_helper::write(filename, "XYZ", dims, XYZ1);

    dims[3] = 1; dims[2] = param_hvec.TE_us.size();
    h5_helper::write(filename, "T", dims, T);

    dims[0] = config.get_scales().size(); dims[1] = 1; dims[2] = 1; dims[3] = 1;
    h5_helper::write(filename, "scales", dims, config.get_scales());

    std::vector<float> TE_us;
    for(int i=0; i<param_hvec.TE_us.size(); i++) TE_us.push_back(param_hvec.TE_us[i]*param.timestep_us*1e-6); 
    dims[0] = TE_us.size(); dims[1] = 1; dims[2] = 1; dims[3] = 1;
    h5_helper::write(filename, "TE", dims, TE_us);
}

bool monte_carlo::run(std::string config_filename) // simulation_parameters param, std::map<std::string, std::vector<std::string> > filenames, std::vector<double> scale
{   
    auto start_run = std::chrono::high_resolution_clock::now();
    // ========== read config file ==========
    if(config.prepare(config_filename, &param, &param_hvec) == false)
        return false; 
    if (param.prepare(param_hvec) == false)
        return false;
    allocate_memory();
    
    // BOOST_LOG_TRIVIAL(info) << "\n" << std::string(20, '-') << "\nSimulation parameters:\n" << param.dump() << "\n" << std::string(20, '-') ;
    size_t trj  = param.enRecordTrajectory ? param.n_timepoints * (param.n_dummy_scan + 1) : 1;
    size_t ind_fieldmap = 0;
    std::vector<float> gradient_mTm_orig = param_hvec.gradient_mTm;

    param_uvec.copy_from_host(param_hvec);
#ifdef __HIPCC__
    if (gpu_disabled == false) { 
        param_dvec.copy_from_host(param_hvec);
        param_uvec.copy_from_device(param_dvec);
    }
#endif
    for (auto &file_phantom : config.get_filename("PHANTOM")){
        BOOST_LOG_TRIVIAL(info) << "Simulating phantom: " << file_phantom;
        if(read_phantom(file_phantom) == false)
            return false;        
        if(initialize_position(config.get_filename("XYZ0")[ind_fieldmap], param.seed) == false)
            return false;
        if(initialize_magnetization(config.get_filename("M0")[ind_fieldmap]) == false)
            return false;

        XYZ0_scaled      = XYZ0;
        param.matrix_length  = mask.size(); // update the matrix length based on the mask size from the recent read
        param.fieldmap_exist = fieldmap.size() > 0;
       
        // convert fieldmap from T to degree per timestep
        float Tesla2deg_pertimestep = param.B0 * param.timestep_us * 1e-6 * GAMMA * RAD2DEG;
        BOOST_LOG_TRIVIAL(info) << "Conversion factor from T to degree per timestep: " << Tesla2deg_pertimestep;
        if(param.fieldmap_exist) 
            std::transform(std::execution::par_unseq, fieldmap.begin(), fieldmap.end(), fieldmap.begin(), [Tesla2deg_pertimestep](auto x) { return x*Tesla2deg_pertimestep;});

        // ========== move to GPU memory ========== 
#ifdef __HIPCC__
        if (gpu_disabled == false) { 
            BOOST_LOG_TRIVIAL(info) << "Moving data to GPU memory.";
            // calculate required memory and avaialbe memory 
            if (check_memory_size(get_total_memory()) == false)
                return false; 
            d_fieldmap = fieldmap;
            d_XYZ0 = XYZ0;
            d_XYZ1 = XYZ1;
            d_mask = mask;
            d_M0 = M0;
            d_M1 = M1;
            d_T = T;
        }
#endif

        for (int i = 0; i < 3; i++) // FoV scaling
            param.fov[i] = fov[i]; 
        // ========== run ==========   
        uint32_t ind_scale = 0;
        std::vector<uint32_t> v(param.n_spins);
        std::generate(std::execution::seq, v.begin(), v.end(), [n = 0] () mutable { return n++; }); 
        
        auto start_sim = std::chrono::high_resolution_clock::now();
        auto bar = barkeep::ProgressBar(&ind_scale, {.total = param.n_scales, .message = "Simulating", .style = barkeep::ProgressBarStyle::Rich,});
        for (const auto scale : config.get_scales())
        {   
            BOOST_LOG_TRIVIAL(info) << "Simulating scale " << scale; 
            // FoV scaling
            if(config.get_scale_type() == e_scale_type::s_fov){           
                std::transform(std::execution::par_unseq, XYZ0.begin(), XYZ0.end(), XYZ0_scaled.begin(), [scale](auto& c){return c*scale;}); 
                for (int i = 0; i < 3; i++) // FoV scaling
                    param.fov[i] = scale * fov[i]; 
#ifdef __HIPCC__
                if (gpu_disabled == false)
                    d_XYZ0 = XYZ0_scaled;
#endif
             } 
             // Gradient scaling
             else if (config.get_scale_type() == e_scale_type::s_gradient){                                
                std::transform(std::execution::par_unseq, gradient_mTm_orig.begin(), gradient_mTm_orig.end(), param_hvec.gradient_mTm.begin(), [scale](auto& c){return c*scale;}); 
#ifdef __HIPCC__
                if (gpu_disabled == false){
                    param_dvec.gradient_mTm = param_hvec.gradient_mTm;
                    param_uvec.gradient_mTm.ptr = thrust::raw_pointer_cast(param_dvec.gradient_mTm.data());
                }
#endif
             }

            // here we need to check voxel size and step size to make sure that the simulation is stable: doi:10.1016/j.neuroimage.2018.06.046 & https://submissions.mirasmart.com/ISMRM2024/Itinerary/PresentationDetail.aspx?evdid=4684
            
            // ========== simulation kernel  ==========
#ifdef __HIPCC__
            if(gpu_disabled){
#endif           
                std::for_each(std::execution::par_unseq, v.begin(), v.end(), [&](int spin) {sim(param, param_uvec, 
                                                                                            fieldmap.data(), 
                                                                                            mask.data(), 
                                                                                            M0.data(), 
                                                                                            XYZ0_scaled.data(), 
                                                                                            M1.data() + 3*param_hvec.TE_us.size()*param.n_spins*ind_scale, 
                                                                                            XYZ1.data() + 3*param.n_spins*trj*ind_scale, 
                                                                                            T.data() + param_hvec.TE_us.size()*param.n_spins*ind_scale,
                                                                                            spin);});
#ifdef __HIPCC__  
            }else{         
                size_t numGrid = (param.n_spins + BLOCKS - 1) / BLOCKS;
                cu_sim<<<numGrid, BLOCKS, 0>>>(param, param_uvec, 
                                                thrust::raw_pointer_cast(d_fieldmap.data()), 
                                                thrust::raw_pointer_cast(d_mask.data()),
                                                thrust::raw_pointer_cast(d_M0.data()),
                                                thrust::raw_pointer_cast(d_XYZ0.data()),
                                                thrust::raw_pointer_cast(d_M1.data() + 3*param_dvec.TE_us.size()*param.n_spins*ind_scale),
                                                thrust::raw_pointer_cast(d_XYZ1.data() + 3*param.n_spins*trj*ind_scale),
                                                thrust::raw_pointer_cast(d_T.data() + param_dvec.TE_us.size()*param.n_spins*ind_scale));
                gpuCheckKernelExecutionError(__FILE__, __LINE__);
            }
#endif     
            ind_scale++;       
        }        
        bar->done();

        auto elapsed_sim = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start_sim).count() / 1000.0;
        int precision = elapsed_sim>10 ? 0 : (elapsed_sim > 1 ? 1 : 3);
        BOOST_LOG_TRIVIAL(info) << "Simulation took " << std::fixed << std::setprecision(precision) <<  elapsed_sim << " seconds.";

        // ========== save results ========== 
        BOOST_LOG_TRIVIAL(info) << "Saving the results to disk.";
        save(config.get_output_filename(ind_fieldmap));
        ind_fieldmap++;
    }

    auto elapsed_run = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start_run).count() / 1000.0;
    int precision = elapsed_run>10 ? 0 : (elapsed_run > 1 ? 1 : 3);
    BOOST_LOG_TRIVIAL(info) << "Entire run took " << std::fixed << std::setprecision(precision) <<  elapsed_run << " seconds.";
    return true;
} 

} // namespace sim