/* --------------------------------------------------------------------------
 * Project: SpinWalk
 * File: spinwalk.cu
 *
 * Author   : Ali Aghaeifar <ali.aghaeifar@tuebingen.mpg.de>
 * Date     : 10.02.2023
 * Descrip  : simulating BOLD in microvascular network
 * -------------------------------------------------------------------------- */

// compile(lin) :  nvcc microvascular_gpu.cu -Xptxas -v -O3  -arch=compute_86 -code=sm_86  -Xcompiler -fopenmp -o sim_microvascular
// compile(win) :  nvcc microvascular_gpu.cu -Xptxas -v -O3  -arch=compute_86 -code=sm_86  -Xcompiler /openmp -std=c++17 -o sim_microvascular

#include <random>
#include <filesystem>
#include "hip/hip_runtime_api.h"
#include "kernels.cuh"
#include "file_utils.h"
#include "tqdm.h"

#define THREADS_PER_BLOCK  64

using namespace std;



bool simulate(simulation_parameters param, std::map<std::string, std::vector<std::string> > filenames, std::vector<float> sample_length_scales)
{
    std::vector<float> fieldmap;
    std::vector<char> mask;
    // ========== checking GPU(s) ==========
    int32_t device_count;
    checkCudaErrors(hipGetDeviceCount(&device_count));

    param.n_spins /= device_count; // spins will be distributed in multiple GPUs (if there is). We hope it is divisible 
    int32_t numBlocks = (param.n_spins + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    uint32_t len0 = 3 * param.n_spins * device_count;
    uint32_t len1 = len0 * param.n_sample_length_scales;
    uint32_t len2 = len1 * param.n_TE;
    std::vector<float> XYZ0(len0, 0.f); // memory layout(column-wise): [3 x n_spins]
    std::vector<float> XYZ1(len1, 0.f); // memory layout(column-wise): [3 x n_spins x n_sample_length_scales]
    std::vector<float> M0(len0, 0.f);   // memory layout(column-wise): [3 x n_spins]
    std::vector<float> M1(len2, 0.f);   // memory layout(column-wise): [3 x n_TE x n_spins x n_sample_length_scales]

    std::cout << std::string(50, '=') << std::endl;
    for (int16_t fieldmap_no=0; fieldmap_no<param.n_fieldmaps; fieldmap_no++)
    {
        bool hasXYZ0 = false;
        // ========== load files (field-maps, xyz0, m0) ==========
        if(file_utils::read_fieldmap(filenames.at("fieldmap")[fieldmap_no], fieldmap, mask, param) == false)
            return false;

        if(filenames.at("xyz0")[fieldmap_no].empty() == false)
        {
            if(file_utils::read_file(filenames.at("xyz0")[fieldmap_no], XYZ0) == false)
                return false;
            
            std::cout << "Checking XYZ0 is not in the mask..." << std::endl;
            uint32_t t = is_masked(XYZ0, mask, &param);
            if(t>0)
            {
                std::cout << ERR_MSG << t << " element(s) of XYZ0 is in the mask. Aborting...!" << std::endl;
                return 1;
            }
            hasXYZ0 = true;
        }

        if(filenames.at("m0")[fieldmap_no].empty() == false)
        {
            if(file_utils::read_file(filenames.at("m0")[fieldmap_no], M0) == false)
                return false;
        }
        else
        {   // all spins are aligned with B0 (M0 = (0, 0, 1))
            long index = 0;
            std::cout << "Generating M0(0, 0, 1)..." << std::endl;
            std::generate(M0.begin(), M0.end(), [&index](){return (index++ % 3 == 2) ? 1.f : 0.f;});
        }

        for(int i=0; i<M0.size()/3 && param.enDebug; i += M0.size()/3/2)
            std::cout << "M0 of the spin " << i << " = (" << M0[3*i] << ", " << M0[3*i+1] << ", " << M0[3*i+2] << ")" << std::endl;

        for(int i=0; i<3; i++)
            param.scale2grid[i] = (param.fieldmap_size[i] - 1.) / param.sample_length[i];
        
        if (hasXYZ0 && param.n_sample_length_scales > 1)
        {
            std::cout << ERR_MSG << "loading XYZ0 from file while having more than 1 sample length scales is not supported!" << std::endl;
            return false;
        }

        // ========== distributing between devices ==========
        std::vector<float *> d_pFieldMap(device_count, NULL);
        std::vector<float *> d_M0(device_count, NULL), d_M1(device_count, NULL);
        std::vector<float *> d_XYZ1(device_count, NULL), d_XYZ0(device_count, NULL), d_XYZ0_scaled(device_count, NULL);
        std::vector<bool *>  d_pMask(device_count, NULL);
        std::vector<simulation_parameters *> d_param(device_count, NULL);
        std::vector<hipStream_t> streams(device_count, NULL);

        #pragma omp parallel for
        for(int32_t d=0; d<device_count; d++)
        {
            checkCudaErrors(hipSetDevice(d));            
            checkCudaErrors(hipStreamCreate(&streams[d]));

            checkCudaErrors(hipMalloc((void**)&d_param[d],         sizeof(simulation_parameters)));
            checkCudaErrors(hipMalloc((void**)&d_pFieldMap[d],     sizeof(fieldmap[0]) * fieldmap.size()));   
            checkCudaErrors(hipMalloc((void**)&d_pMask[d],         sizeof(mask[0]) * mask.size())); 
            checkCudaErrors(hipMalloc((void**)&d_XYZ0[d],          sizeof(float) * 3 * param.n_spins));
            checkCudaErrors(hipMalloc((void**)&d_XYZ0_scaled[d],   sizeof(float) * 3 * param.n_spins));
            checkCudaErrors(hipMalloc((void**)&d_XYZ1[d],          sizeof(float) * 3 * param.n_spins));
            checkCudaErrors(hipMalloc((void**)&d_M0[d],            sizeof(float) * 3 * param.n_spins));
            checkCudaErrors(hipMalloc((void**)&d_M1[d],            sizeof(float) * 3 * param.n_TE * param.n_spins));
            
            checkCudaErrors(hipMemcpyAsync(d_pFieldMap[d], fieldmap.data(),        fieldmap.size()*sizeof(fieldmap[0]), hipMemcpyHostToDevice, streams[d]));
            checkCudaErrors(hipMemcpyAsync(d_pMask[d],     mask.data(),            mask.size() * sizeof(mask[0]),       hipMemcpyHostToDevice, streams[d]));
            checkCudaErrors(hipMemcpyAsync(d_param[d],     &param,                 sizeof(simulation_parameters),       hipMemcpyHostToDevice, streams[d]));
            checkCudaErrors(hipMemcpyAsync(d_M0[d],        &M0[3*param.n_spins*d], 3*param.n_spins*sizeof(M0[0]),       hipMemcpyHostToDevice, streams[d]));
            
            if(hasXYZ0 == false)
            {   // generate initial spatial position for spins, based on sample_length_ref
                printf("GPU %d) Generating random initial position for spins... ", d);
                cu_randPosGen<<<numBlocks, THREADS_PER_BLOCK, 0, streams[d]>>>(d_XYZ0[d], d_param[d], d_pMask[d]);
                gpuCheckKernelExecutionError( __FILE__, __LINE__);
                printf("Done!\n");
            }
            else // copy initial spatial position and magnetization for spins
                checkCudaErrors(hipMemcpyAsync(d_XYZ0[d], &XYZ0[3*param.n_spins*d], 3*param.n_spins*sizeof(XYZ0[0]), hipMemcpyHostToDevice, streams[d]));      
        }

        // ========== run ==========        
        hipEvent_t start;
        hipEvent_t end;
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&end));
        checkCudaErrors(hipEventRecord(start));
        
        tqdm bar;
        simulation_parameters param_local;
        memcpy(&param_local, &param, sizeof(simulation_parameters));
        for (int32_t sl = 0; sl < param.n_sample_length_scales; sl++)
        {
            for (int i = 0; i < 3; i++)
            {
                param_local.sample_length[i] = sample_length_scales[sl] * param.sample_length[i];
                param_local.scale2grid[i] = (param_local.fieldmap_size[i] - 1.) / param_local.sample_length[i];
            }
            
            #pragma omp parallel for
            for (int32_t d = 0; d < device_count; d++)
            {                    
                checkCudaErrors(hipSetDevice(d));
                hipMemcpy(d_param[d], &param_local, sizeof(simulation_parameters), hipMemcpyHostToDevice);

                cu_scalePos<<<numBlocks, THREADS_PER_BLOCK, 0, streams[d]>>>(d_XYZ0_scaled[d], d_XYZ0[d], sample_length_scales[sl], param.n_spins);
                gpuCheckKernelExecutionError(__FILE__, __LINE__);

                cu_sim<<<numBlocks, THREADS_PER_BLOCK, 0, streams[d]>>>(d_param[d], d_pFieldMap[d], d_pMask[d], d_M0[d], d_XYZ0_scaled[d], d_M1[d], d_XYZ1[d]);
                gpuCheckKernelExecutionError(__FILE__, __LINE__);

                int shift = 3*param.n_TE*param.n_spins*device_count*sl + 3*param.n_TE*param.n_spins*d;
                checkCudaErrors(hipMemcpyAsync(M1.data()   + shift, d_M1[d]  , 3*param.n_TE*param.n_spins*sizeof(float), hipMemcpyDeviceToHost, streams[d]));
                shift = 3*param.n_spins*device_count*sl + 3*param.n_spins*d;
                checkCudaErrors(hipMemcpyAsync(XYZ1.data() + shift, d_XYZ1[d], 3*param.n_spins*sizeof(float), hipMemcpyDeviceToHost, streams[d]));
            }
            bar.progress(sl, param.n_sample_length_scales);
        }
        bar.finish();

        float elapsedTime;
        checkCudaErrors(hipEventRecord(end));
        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, end));
        std::cout << "Entire simulation over " << device_count << " GPU(s) took " << std::fixed << std::setprecision(2) << elapsedTime/1000. << " second(s)" << std::endl;

        // ========== clean up GPU ==========
        #pragma omp parallel for
        for(int32_t d=0; d<device_count; d++)
        {
            checkCudaErrors(hipSetDevice(d));   
            checkCudaErrors(hipFree(d_param[d]));
            checkCudaErrors(hipFree(d_pFieldMap[d]));
            checkCudaErrors(hipFree(d_pMask[d]));
            checkCudaErrors(hipFree(d_XYZ0[d]));
            checkCudaErrors(hipFree(d_XYZ0_scaled[d]));
            checkCudaErrors(hipFree(d_M1[d]));
            checkCudaErrors(hipFree(d_XYZ1[d]));
            checkCudaErrors(hipStreamDestroy(streams[d]));            
        }
        checkCudaErrors(hipEventDestroy(start));
        checkCudaErrors(hipEventDestroy(end));
        
        // ========== save results ========== 
        output_header hdr(3, param.n_TE, param.n_spins * device_count, param.n_sample_length_scales);
        file_utils::save_output(M1, filenames.at("m1")[fieldmap_no], hdr, sample_length_scales);

        hdr.dim2 = 1;
        if(filenames.at("xyz1")[fieldmap_no].empty() == false) // do not save if filename is empty
            file_utils::save_output(XYZ1, filenames.at("xyz1")[fieldmap_no], hdr, sample_length_scales);

        std::cout << std::string(50, '=') << std::endl;
    }
    return true;
}


int main(int argc, char * argv[])
{
    // ========== parse command line arguments ==========
    std::vector<std::string> config_files;    
    bool bVerbose = false, bHelp = false;
    for(uint8_t i=1; i<argc; i++)
    {
        if (strcmp(argv[i], "-v") == 0)
            bVerbose = true;
        else if (strcmp(argv[i], "-h") == 0)
            bHelp = true;
        else
            config_files.push_back(argv[i]);
    }

    // ========== print help ==========
    if(argc < 2 || bHelp || config_files.size() == 0)
    {
        std::cout << "Usage: " << argv[0] << " -options <config_file1> <config_file2> ... <config_filen>" << std::endl;
        std::cout << "Options:" << std::endl;
        std::cout << "  -v: verbose" << std::endl;  
        std::cout << "  -h: help (this menu)" << std::endl;      
        print_device_info();
        return 1;
    }

    std::cout << "Running " << config_files.size() << " simulation(s)..." << std::endl;
    for(uint8_t cnf=0; cnf<config_files.size(); cnf++)
    {
        map<string, vector<string> > filenames = {{"fieldmap", 	vector<string>()},  // input:  map of off-resonance in Tesla
                                                  {"xyz0", 		vector<string>()},  // input:  spins starting spatial positions in meters
                                                  {"xyz1", 		vector<string>()},  // output: spins last spatial positions in meters
                                                  {"m0", 		vector<string>()},  // input:  spins initial magnetization
                                                  {"m1", 		vector<string>()}}; // output: spins final magnetization

        std::vector<float> sample_length_scales;
        simulation_parameters param;

        // ========== read config file ==========
        param.fieldmap_size[0] = param.fieldmap_size[1] = param.fieldmap_size[2] = 0;
        param.sample_length[0] = param.sample_length[1] = param.sample_length[2] = 0.f;
        if(file_utils::read_config(config_files[cnf], param, sample_length_scales, filenames) == false)
        {
            std::cout << ERR_MSG << "reading config file failed. Aborting...!" << std::endl;
            return 1;
        }

        if (param.seed == 0)
            param.seed = std::random_device{}();

        param.n_timepoints = param.TR / param.dt; // includes start point

        // ========== Dump Settings ==========
        if(param.enDebug = bVerbose)
        {
            std::cout << "Dumping settings:" << std::endl;
            for (std::map<std::string, std::vector<std::string>>::iterator it=filenames.begin(); it!=filenames.end(); ++it)
                for (int i = 0; i< it->second.size(); i++)
                    std::cout << it->first << "[" << i << "] = " << it->second.at(i) << std::endl;
            
            std::cout << "\nSample length scale = [ ";
            for (int32_t i = 0; i < param.n_sample_length_scales; i++)
                std::cout << sample_length_scales[i] << ", ";
            std::cout << "\r\r ]\n" << std::endl;

            input_header hdr_in;
            if(file_utils::read_header(filenames.at("fieldmap")[0], hdr_in) == false)
                return 1;
            std::copy(hdr_in.fieldmap_size, hdr_in.fieldmap_size+3, param.fieldmap_size);
            std::copy(hdr_in.sample_length, hdr_in.sample_length+3, param.sample_length);
            param.dump();
            std::cout<< std::string(50, '=')  << std::endl;
        }

        if(simulate(param, filenames, sample_length_scales) == false)
            return 1;
    }
    std::cout << "Simulation(s) finished successfully!" << std::endl;
    return 0;
}
