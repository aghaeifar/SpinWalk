#include "hip/hip_runtime.h"

/* --------------------------------------------------------------------------
 * Project: SpinWalk
 * File: arbitrary_gradient.h
 *
 * Author   : Ali Aghaeifar <ali.aghaeifar@tuebingen.mpg.de>
 * Date     : 15.05.2024
 * Descrip  : 
 * -------------------------------------------------------------------------- */
#include <highfive/highfive.hpp>
#include <filesystem>
#include <random>
#include "indicators.hpp"
#include "cylinder.cuh"
#include "basic_functions.cuh"

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

using namespace indicators;

cylinder::cylinder()
{
    m_radius = 0;
    m_orientation = 0;
}

cylinder::cylinder(float fov_um, size_t resolution, float dChi, float Y, float radius_um, float BVF, float orientation, int32_t seed, std::string filename)
: shape(fov_um, resolution, dChi, Y, BVF, seed, filename)
{
    set_cylinder_parameters(radius_um, orientation);
}

cylinder::~cylinder()
{

}

void cylinder::set_cylinder_parameters(float radius, float orientation)
{
    m_radius = radius;
    m_orientation = orientation;
    float B0_orig[3] = {0.f, 0.f, 1.f};
    yrot(orientation, B0_orig, B0);
}

void cylinder::generate_shapes()
{
    if(2*m_radius>=m_fov)
    {
        std::cerr << "Error: The radius of the cylinder is too large for the given FOV!\n";
        return;
    }
    std::cout << "Generating coordinates...for target BVF = " << m_BVF << "% ...\n"; 
    bool is_random_radius = m_radius < 0;
    float max_radius    = m_radius>0 ? m_radius:-m_radius;
    m_cylinder_points.clear();
    m_cylinder_radii.clear();
    float cyl_pnt[3], cyl_rad ;
    float curr_BVF = 0;
    // srandom engine
    std::mt19937 gen(m_seed); // Mersenne Twister generator
    std::uniform_real_distribution<float> dist(0.f, 1.f); 
      
    float distance, vol_cyl = 0, vol_cyl_total = 0, vol_tol = m_fov*m_fov*m_fov;
    auto start = std::chrono::high_resolution_clock::now();
    while(curr_BVF < m_BVF)
    {
        cyl_rad = is_random_radius ? dist(gen) * max_radius : max_radius;
        for (size_t i = 0; i < 3; i++) // generate a random point for a sphere which fit in the FOV
            cyl_pnt[i] = dist(gen) * (m_fov+2*cyl_rad) - cyl_rad;   
        
        // check if sphere coordinate is ok
        size_t c;
        for (c=0; c<m_cylinder_radii.size(); c++)
        {   
            float p2p1[3];
            subtract(cyl_pnt, m_cylinder_points[c].data(), p2p1);
            distance = sqrtf(p2p1[0]*p2p1[0] + p2p1[1]*p2p1[1]);
            // if the sphere is inside another sphere, generate a new sphere
            if(distance <= m_cylinder_radii[c] ||  distance <= cyl_rad)
                break;
            // adjust the radius of the sphere to avoid overlap
            if (distance < m_cylinder_radii[c] + cyl_rad)
            {
                if (!is_random_radius)
                    break;            
                cyl_rad = distance - m_cylinder_radii[c];
            }
        }
        if (c < m_cylinder_radii.size())
            continue;

        vol_cyl = calculate_volume(cyl_pnt, cyl_rad);
        // if the total volume of the cylinders is more than the target BVF or the cylinder is outside of volume, skip this cylinder
        if (100*(vol_cyl + vol_cyl_total) / vol_tol > 1.02*m_BVF || vol_cyl < 0)
            continue;
        
        vol_cyl_total += vol_cyl;
        curr_BVF = 100.*vol_cyl_total/vol_tol; 
        m_cylinder_points.push_back({cyl_pnt[0], cyl_pnt[1], cyl_pnt[2]});
        m_cylinder_radii.push_back(cyl_rad);     
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::cout << m_cylinder_radii.size() << " coordinates generated successfully! Elapsed Time: " << std::chrono::duration_cast<std::chrono::seconds>(end - start).count() << " s" << std::endl;
}


float cylinder::calculate_volume(float *cyl_pnt, float cyl_rad)
{
    bool intersect = false;
    // check if the cylinder is completely outside the volume
    for (size_t i = 0; i < 2; i++)
        if(cyl_pnt[i]+cyl_rad < 0 || cyl_pnt[i]-cyl_rad > m_fov)
            return -1.f;
    // check if the cylinder is completely inside the volume
    for (size_t i = 0; i < 2; i++) // only x and y directions
        if (cyl_pnt[i] < cyl_rad - 1.5 || cyl_pnt[i] > m_fov - cyl_rad + 1.5) // 1.5 is a small margin because of a possible larger volume after discretization
            intersect = true;

    if (intersect == false)
        return M_PI * cyl_rad*cyl_rad * m_fov;    

    // find the bounding box of the sphere
    float v_size = m_fov / m_resolution;
    float cyl_rad2  = cyl_rad*cyl_rad;
    size_t res1 = m_resolution;
    size_t res2 = res1 * res1;
    int32_t cyl_pnt_vox[3] = {int32_t(cyl_pnt[0]/v_size), int32_t(cyl_pnt[1]/v_size), int32_t(cyl_pnt[2]/v_size)};
    int32_t cyl_rad_vox = std::ceil(cyl_rad / m_fov * m_resolution)+1; 
    int32_t z_min = 0;
    int32_t z_max = m_resolution;
    int32_t x_min = std::max(0, cyl_pnt_vox[0] - cyl_rad_vox);
    int32_t x_max = std::min((int32_t)m_resolution, cyl_pnt_vox[0] + cyl_rad_vox + 2);
    int32_t y_min = std::max(0, cyl_pnt_vox[1] - cyl_rad_vox);
    int32_t y_max = std::min((int32_t)m_resolution, cyl_pnt_vox[1] + cyl_rad_vox + 2);
    int32_t counter = 0;
    #pragma omp parallel for
    for(int32_t pz=z_min; pz<z_max; pz++)
    for(int32_t py=y_min; py<y_max; py++)
    for(int32_t px=x_min; px<x_max; px++)
    {
        size_t p = px*res2 + py*res1 + pz;
        float *grid = &m_grid[3*p];
        float p2p1[3];
        // distance between the points and vessel axis and vector from the projection point to the point
        subtract(grid, cyl_pnt, p2p1);  // vector from the spatial points to the cylinder point
        float distance2 = p2p1[0]*p2p1[0] + p2p1[1]*p2p1[1];   // distance^2 between the points and vessel axis. this is distance in the plane perpendicular to the vessel axis
        if (distance2 <= cyl_rad2)
        {
            #pragma omp atomic
            ++counter;
        }
    }
    return counter * v_size*v_size*v_size ;
}


void cylinder::generate_mask_fieldmap()
{   
    // set the cylinders orientation if they are parallel
    std::cout << "Generating cylinders..." << std::endl;    
    size_t res1 = m_resolution;
    size_t res2 = res1 * res1;
    size_t res3 = res1 * res2;
    int32_t x_min, x_max, y_min, y_max, z_min, z_max;
    int32_t cyl_rad_vox, cyl_pnt_vox[3];

    std::cout<<"B0 direction: ["<<B0[0]<<", "<<B0[1]<<", "<<B0[2]<<"]\n";
    std::cout<<"Allocating memory..."<<std::endl;
    m_fieldmap.resize(m_calc_fieldmap ? res3:0, 0.f);
    m_mask.resize(res3, 0);
    float v_size = m_fov / m_resolution;

    float cyl_dir[3] = {0.0, 0.0, 1.0};
    float theta_c, theta_c2, theta_s2, B0_prj[3] = {B0[0], B0[1], 0.0};     // project B0 from the projection point to the point to the plane perpendicular to the vessel axis  
    normalize(B0_prj);

    theta_c  = cos(m_orientation * M_PI / 180); // cos(theta), angle between axis of vessel and B0 (in radian)
    theta_c2 = theta_c * theta_c;
    theta_s2 = 1. - theta_c2; // sin^2(theta)
    
    ProgressBar bar{option::ShowPercentage{true}, option::Start{"["}, option::Fill{"="}, option::Lead{">"}, option::End{"]"}};
    auto start = std::chrono::high_resolution_clock::now();
    for (size_t c = 0; c < m_cylinder_radii.size(); c++)
    {
        float *cyl_pnt  = m_cylinder_points[c].data();
        float cyl_rad   = m_cylinder_radii[c];
        float cyl_rad2  = cyl_rad*cyl_rad;
        cyl_rad_vox = std::ceil(cyl_rad / v_size)+1;
        cyl_pnt_vox[0] = int32_t(cyl_pnt[0]/v_size); 
        cyl_pnt_vox[1] = int32_t(cyl_pnt[1]/v_size);
        cyl_pnt_vox[2] = int32_t(cyl_pnt[2]/v_size);

        // find the bounding box of the sphere
        z_min = 0;
        z_max = m_resolution;
        if (m_calc_fieldmap)
        {
            x_min = std::max(0, cyl_pnt_vox[0] - cyl_rad_vox*8);
            x_max = std::min((int32_t)m_resolution, cyl_pnt_vox[0] + cyl_rad_vox*10);
            y_min = std::max(0, cyl_pnt_vox[1] - cyl_rad_vox*8);
            y_max = std::min((int32_t)m_resolution, cyl_pnt_vox[1] + cyl_rad_vox*10);
        } else 
        {
            x_min = std::max(0, cyl_pnt_vox[0] - cyl_rad_vox);
            x_max = std::min((int32_t)m_resolution, cyl_pnt_vox[0] + cyl_rad_vox + 2);
            y_min = std::max(0, cyl_pnt_vox[1] - cyl_rad_vox);
            y_max = std::min((int32_t)m_resolution, cyl_pnt_vox[1] + cyl_rad_vox + 2);
        }

        #pragma omp parallel for
        for(int32_t pz=z_min; pz<z_max; pz++)
        for(int32_t py=y_min; py<y_max; py++)
        for(int32_t px=x_min; px<x_max; px++)
        {
            size_t p = px*res2 + py*res1 + pz;
            float *grid = &m_grid[3*p];
            float p2p1[3], temp[3], perpendicular[3], distance2, phi_c, phi_2c2_1 ;
            // distance between the points and vessel axis and vector from the projection point to the point
            subtract(grid, cyl_pnt, p2p1);  // vector from the spatial points to the cylinder point
            distance2 = p2p1[0]*p2p1[0] + p2p1[1]*p2p1[1];   // distance^2 between the points and vessel axis. this is distance in the plane perpendicular to the vessel axis
            if (distance2 <= cyl_rad2)
                m_mask[p] = 1;

            if (m_calc_fieldmap)
            {
                multiply(dot_product(cyl_dir, p2p1), cyl_dir, temp); // project vector temp onto the cylinder direction vector
                add(temp, cyl_pnt, temp);               // projection point
                subtract(grid, temp, perpendicular);    // vector from the spatial points to the cylinder axis
                // angle between the projected B0 and the vector from the projection point to the point
                phi_c = dot_product(perpendicular, B0_prj) / norm(perpendicular); // cos(phi)
                phi_2c2_1 = 2 * phi_c * phi_c - 1;      // cos(2*phi)
                // calculate the fieldmap from the vessel 
                if (distance2 > cyl_rad2)  // outside the cylinder              
                    m_fieldmap[p] += 2*M_PI * (1-m_Y)*m_dChi * (cyl_rad2 / distance2) * phi_2c2_1 * theta_s2;   
                else // inside the cylinder                
                    m_fieldmap[p] += 2*M_PI * (1-m_Y)*m_dChi * (theta_c2 - 1/3);
            }                 
        }        
        bar.set_progress(100 * (c+1)/float(m_cylinder_radii.size()));
    } 
    
    m_BVF = std::accumulate(m_mask.begin(), m_mask.end(), 0) * 100.0 / m_mask.size();
    std::cout << "Actual BVF = " << m_BVF << "% ...\n";   
    auto end = std::chrono::high_resolution_clock::now();
    std::cout << "Cylinders generated successfully! " << "Elapsed Time: " << std::chrono::duration_cast<std::chrono::seconds>(end - start).count() << " s\n";
}

void cylinder::print_info()
{
    shape::print_info();
    std::cout << "  Radius: " << m_radius << " um\n";
    std::cout << "  BVF: " << m_BVF << "\n";
    std::cout << "  Orientation: " << m_orientation << " rad\n";
}

// -------------------------------------------------------------------------- //

bool cylinder::run()
{
    print_info();
    create_grid();
    generate_shapes();
    generate_mask_fieldmap();
    save();
    return true;
}