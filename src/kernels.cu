#include "hip/hip_runtime.h"
/* --------------------------------------------------------------------------
 * Project: SpinWalk
 * File: kernels.cuh
 *
 * Author   : Ali Aghaeifar <ali.aghaeifar@tuebingen.mpg.de>
 * Date     : 10.02.2023
 * Descrip  : simulating BOLD in microvascular network
 * -------------------------------------------------------------------------- */


#include <algorithm>
#include <thrust/random.h>
#include "kernels.cuh"
#include "rotation.cuh"
#include "hip/hip_runtime_api.h"


//---------------------------------------------------------------------------------------------
//  
//---------------------------------------------------------------------------------------------

__device__ __forceinline__ void dephase_relax(float *m0, float *m1, float accumulated_phase, float T1, float T2, float time_elapsed)
{
    // dephase                
    zrot(accumulated_phase, m0, m1); 
    // relax
    relax(exp(-time_elapsed/T1), exp(-time_elapsed/T2), m1);
}

__global__ void cu_sim(const simulation_parameters *param, const float *pFieldMap, const uint8_t *pMask, const float *M0, const float *XYZ0, float *M1, float *XYZ1)
{
    auto spin_no = blockIdx.x * blockDim.x + threadIdx.x ;
    if (spin_no >= param->n_spins)
        return;
    
    thrust::minstd_rand gen(param->seed + spin_no);
    thrust::normal_distribution<float> dist_random_walk_xyz(0.f, sqrt(6 * param->diffusion_const * param->dt));
    //thrust::uniform_real_distribution<float> dist_random_walk_xyz(-sqrt(6 * param.diffusion_const * param.dt), sqrt(6 * param.diffusion_const * param.dt));
    gen.discard(param->seed + spin_no); // each spins has its own seed, and param->seed differes for each GPU in HPC with multiple GPUs

    //uint16_t n_timepoints_local;
    float field = 0., T1=0., T2=0., rf_phase = param->RF_PH[0], time_elapsed = 0.; 
    float m0[3], m1[3]; 
    float xyz[3], xyz_new[3];
    for(uint32_t i=0, shift=3*spin_no; i<3; i++)
    {
        xyz[i] = XYZ0[shift + i];
        m0[i]  = M0[shift + i];
    }
    
    bool is_lastscan = false;
    for (uint32_t dummy_scan = 0; dummy_scan < param->n_dummy_scan + 1; dummy_scan++)
    {
        is_lastscan = (dummy_scan == param->n_dummy_scan);
        
        while (rf_phase > 360.0)
            rf_phase -= 360.0;
        while (rf_phase < 0)
            rf_phase += 360.0;
        
        // ------ apply the first RF pulse. The start time for the first RF pulse is always 0 ------
        xrot_withphase (param->s, param->c, rf_phase += param->phase_cycling, m0, m1);

        for(uint8_t i=0; i<3; i++) // copy m1 to m0
            m0[i] = m1[i];

        // ------ loop over timepoints ------
        uint64_t ind=0, ind_old=param->matrix_length+1;
        uint32_t current_timepoint = 0, old_timepoint = 0;
        uint16_t current_rf = 1, current_te = 0, counter_dephasing = 0, counter_gradient = 0;
        float accumulated_phase = 0.f;        
        while (current_timepoint < param->n_timepoints) // param->n_timepoints is the total number of timepoints (= TR/dwelltime)
        {
            // ------ generate random walks and wrap around the boundries ------
            float rnd_wlk;
            for (uint8_t i=0; i<3; i++)
            {
                rnd_wlk = dist_random_walk_xyz(gen);
                xyz_new[i] = xyz[i] + rnd_wlk; // new spin position after random-walk
                if (xyz_new[i] < 0)
                    xyz_new[i] += param->enCrossBoundry ? param->sample_length[i] : -2*rnd_wlk; // rnd_wlk is negative here
                else if (xyz_new[i] > param->sample_length[i])
                    xyz_new[i] -= param->enCrossBoundry ? param->sample_length[i] : 2*rnd_wlk;
            }
            
            // ------ subscripts to linear indices ------
            ind = sub2ind(ROUND(xyz_new[0]*param->scale2grid[0]+1.), ROUND(xyz_new[1]*param->scale2grid[1]+1.), ROUND(xyz_new[2]*param->scale2grid[2]+1.), param->fieldmap_size[0], param->fieldmap_size[1]);
            
            // ------ accumulate phase ------
            if(ind != ind_old) // used this trick for fewer access to the global memory which is slow. Helpful for large samples!
            {               
                if (pMask[ind] != 0 && param->enMultiTissue == false) // check doesn't cross a vessel 
                    continue;       
                field = pFieldMap[ind_old = ind];
                ind = pMask[ind]; // the index of the tissue type
                T1 = param->T1[ind];
                T2 = param->T2[ind];
            }     
            accumulated_phase += field;

            // ------ apply dephasing if there is any ------
            if(counter_dephasing < param->n_dephasing && param->dephasing_T[counter_dephasing] == current_timepoint)
            {
                accumulated_phase += (float)spin_no * param->dephasing[counter_dephasing] / (float)param->n_spins; // assign dephasing linearly to spins 
                counter_dephasing++;
            }

            // ------ apply gradient if there is any ------
            if(counter_gradient < param->n_gradient && param->gradient_T[counter_gradient] == current_timepoint)
            {
                const float *Gxyz = param->gradient_xyz + 3*counter_gradient;
                accumulated_phase +=  (Gxyz[0]*xyz_new[0] + Gxyz[1]*xyz_new[1] + Gxyz[2]*xyz_new[2]) * param->dt*GAMMA*RAD2DEG; //  Gx * x + Gy * y + Gz * z
                counter_gradient++;
            }
                 
            // ------ apply other RF pulse if there is any ------
            if(current_rf < param->n_RF && param->RF_ST[current_rf] == current_timepoint)
            {
                // dephase and relax    
                time_elapsed = (current_timepoint - old_timepoint) * param->dt;
                dephase_relax(m0, m1, accumulated_phase, T1, T2, time_elapsed);
                // apply RF pulse
                xrot_withphase (param->RF_FA[current_rf], param->RF_PH[current_rf], m1, m0); // Note m0 and m1 are swapped here, so that we can use m0 for the next iteration
                accumulated_phase = 0; // reset phase since we have it now applied
                old_timepoint = current_timepoint;
                current_rf++;
            }

            // ------ echoes are only recorded in the last scan ------
            if (is_lastscan && current_te < param->n_TE && param->TE[current_te] == current_timepoint)
            {
                // dephase and relax                
                time_elapsed = (current_timepoint - old_timepoint) * param->dt;
                dephase_relax(m0, m1, accumulated_phase, T1, T2, time_elapsed);
                // save echo and copy m1 to m0 for the next iteration
                for (uint32_t i=0, shift=3*param->n_TE*spin_no + 3*current_te; i<3; i++)
                    M1[shift + i] = m0[i] = m1[i];
                accumulated_phase = 0; // reset phase since we have applied it in the previous step
                old_timepoint = current_timepoint;
                current_te++;
            }

            // update old position with the new one
            for (uint8_t i = 0; i < 3; i++)
                xyz[i] = xyz_new[i];
            // increase timepoint
            current_timepoint++;            
        }
        // dephase and relax    
        time_elapsed = (current_timepoint - old_timepoint) * param->dt;
        dephase_relax(m0, m1, accumulated_phase, T1, T2, time_elapsed);

        // copy m1 to m0 for the next iteration
        for(uint8_t i=0; i<3; i++)
            m0[i] = m1[i];
    }
    // save final position
    for (uint32_t i=0, shift=3*spin_no; i<3; i++)
        XYZ1[shift + i] = xyz[i];
}


//---------------------------------------------------------------------------------------------
//  
//---------------------------------------------------------------------------------------------

__global__ void cu_scalePos(float *scaled_xyz, float *initial_xyz, float scale, uint64_t size)
{
    uint64_t n = blockIdx.x * blockDim.x + threadIdx.x ;
    if(n < size)
    {
        uint64_t ind = 3*n;
        scaled_xyz[ind+0] = initial_xyz[ind+0] * scale;
        scaled_xyz[ind+1] = initial_xyz[ind+1] * scale;
        scaled_xyz[ind+2] = initial_xyz[ind+2] * scale;
    }
}

//---------------------------------------------------------------------------------------------
// CUDA kernel to perform array multiplication with a constant
//---------------------------------------------------------------------------------------------
__global__ void cu_scaleArray(float *array, float scale, uint64_t size)
{
    auto n = blockIdx.x * blockDim.x + threadIdx.x ;
    if(n < size)
        array[n] *= scale;
}

//---------------------------------------------------------------------------------------------
// CUDA kernel to generate random initial position
//---------------------------------------------------------------------------------------------

__global__ void cu_randPosGen(float *spin_position_xyz, simulation_parameters *param, const uint8_t *pMask, uint32_t spin_no)
{
    spin_no = blockIdx.x * blockDim.x + threadIdx.x ;
    if(spin_no >= param->n_spins)
        return;

    thrust::minstd_rand  gen(param->seed + spin_no);
    thrust::uniform_real_distribution<float> dist_initial_point(0., 1.);
    gen.discard(param->seed + spin_no);

    float scale2grid[3];
    for(int i=0; i<3; i++)
        scale2grid[i] = (param->fieldmap_size[i]-1.) / param->sample_length[i];

    uint64_t index = 0;
    float *xyz = spin_position_xyz + 3*spin_no;
    do
    {
        for (uint8_t i = 0; i < 3; i++)
            xyz[i] = dist_initial_point(gen) * param->sample_length[i];
        index = sub2ind(ROUND(xyz[0]*scale2grid[0]+1.), ROUND(xyz[1]*scale2grid[1]+1.), ROUND(xyz[2]*scale2grid[2]+1.), param->fieldmap_size[0], param->fieldmap_size[1]);
    } while (pMask[index] != 0 && param->enMultiTissue == false);
}

//---------------------------------------------------------------------------------------------
//  check for CUDA and GPU device
//---------------------------------------------------------------------------------------------
void print_device_info()
{
    const int kb = 1024;
    const int mb = kb * kb;
    size_t free, total;
    
    int32_t device_count, cuda_version, driver_version;
    checkCudaErrors(hipGetDeviceCount(&device_count));
    hipRuntimeGetVersion(&cuda_version);
    hipDriverGetVersion(&driver_version);
    std::cout << "\nDriver version: "<< driver_version << ", CUDA version: "<< cuda_version << ", Number of devices: " << device_count << std::endl;

    hipDeviceProp_t device_properties;
    for(int i=0; i<device_count; i++)
    {
        hipSetDevice(i);
        hipGetDeviceProperties(&device_properties, i);
        std::cout << "Device " << i+1 << ", " << device_properties.name << std::endl;
        // hipDeviceReset();
        hipMemGetInfo(&free, &total);
        std::cout << "Free GPU memory: " << free / mb << " MB (out of " << total / mb << " MB)" << std::endl;
    }
}