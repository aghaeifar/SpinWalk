#include "hip/hip_runtime.h"
/* --------------------------------------------------------------------------
 * Project: SpinWalk
 * File: kernels.cuh
 *
 * Author   : Ali Aghaeifar <ali.aghaeifar@tuebingen.mpg.de>
 * Date     : 10.02.2023
 * Descrip  : simulating BOLD in microvascular network
 * -------------------------------------------------------------------------- */


#include <algorithm>
#include "kernels.cuh"
#include "rotation.cuh"
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <boost/log/trivial.hpp>
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
//---------------------------------------------------------------------------------------------
//  
//---------------------------------------------------------------------------------------------

uint8_t find_max(const std::vector<uint8_t> &data)
{
    thrust::device_vector<uint8_t> gpu_vec(data.begin(), data.end());
    uint8_t m = *thrust::max_element(gpu_vec.begin(), gpu_vec.end());
    return m;
}

__device__ __forceinline__ void dephase_relax(float *m0, float *m1, float accumulated_phase, float T1, float T2, float time_elapsed)
{
    // dephase                
    zrot(accumulated_phase, m0, m1); 
    // relax
    relax(exp(-time_elapsed/T1), exp(-time_elapsed/T2), m1);
}

__global__ void cu_sim(const simulation_parameters *param, const float *pFieldMap, const uint8_t *pMask, const float *M0, const float *XYZ0, float *M1, float *XYZ1, uint8_t *T)
{
    auto spin_no = blockIdx.x * blockDim.x + threadIdx.x ;
    if (spin_no >= param->n_spins)
        return;

    float *xyz1 = XYZ1 + 3*spin_no * (param->enRecordTrajectory ? (param->n_dummy_scan + 1)*(param->n_timepoints) : 1);

    thrust::minstd_rand gen_r(param->seed + spin_no);
    thrust::minstd_rand gen_u(param->seed + spin_no);
    thrust::normal_distribution<float> dist_random_walk_xyz(0., sqrt(6. * param->diffusion_const * param->dt));
    // thrust::uniform_real_distribution<float> dist_random_walk_xyz(-sqrt(6 * param->diffusion_const * param->dt), sqrt(6 * param->diffusion_const * param->dt));
    thrust::uniform_real_distribution<float> dist_cross_tissue(0.0f, 1.0f);
    gen_r.discard(param->seed + spin_no); // each spins has its own seed, and param->seed differes for each GPU in HPC with multiple GPUs
    gen_u.discard(param->seed + spin_no); // each spins has its own seed, and param->seed differes for each GPU in HPC with multiple GPUs

    uint32_t itr = 0;
    float field = 0., T1=0., T2=0., rf_phase = param->RF_PH[0], time_elapsed = 0.; 
    float m0[3], m1[3]; 
    float xyz_new[3];
    for(uint32_t i=0, shift=3*spin_no; i<3; i++)
    {
        xyz1[i] = XYZ0[shift + i];
        m0[i]  = M0[shift + i];
    }
    // tissue type
    uint8_t ts, ts_old;
    auto indx = sub2ind(xyz1[0]*param->scale2grid[0], xyz1[1]*param->scale2grid[1], xyz1[2]*param->scale2grid[2], param->fieldmap_size[0], param->fieldmap_size[1]);
    ts_old = pMask[indx];

    bool is_lastscan = false;
    for (uint32_t dummy_scan = 0; dummy_scan < param->n_dummy_scan + 1; dummy_scan++)
    {
        is_lastscan = (dummy_scan == param->n_dummy_scan);
        
        while (rf_phase > 360.0)
            rf_phase -= 360.0;
        while (rf_phase < 0)
            rf_phase += 360.0;
        
        // ------ apply the first RF pulse. The start time for the first RF pulse is always 0 ------
        xrot_withphase (param->s, param->c, rf_phase += param->phase_cycling, m0, m1);

        for(uint8_t i=0; i<3; i++) // copy m1 to m0
            m0[i] = m1[i];

        // ------ loop over timepoints ------
        uint64_t ind=0, ind_old=param->matrix_length+1;
        uint32_t current_timepoint = 0, old_timepoint = 0;
        uint16_t current_rf = 1, current_te = 0, counter_dephasing = 0, counter_gradient = 0;
        float accumulated_phase = 0.f;        
        while (current_timepoint < param->n_timepoints) // param->n_timepoints is the total number of timepoints (= TR/dwelltime)
        {
            // ------ generate random walks and wrap around the boundries ------
            float rnd_wlk;
            for (uint8_t i=0; i<3; i++)
            {
                rnd_wlk = dist_random_walk_xyz(gen_r);
                xyz_new[i] = xyz1[i] + rnd_wlk; // new spin position after random-walk
                if (xyz_new[i] < 0)
                    xyz_new[i] += param->enCrossBoundry ? param->sample_length[i] : -2*rnd_wlk; // rnd_wlk is negative here
                else if (xyz_new[i] > param->sample_length[i])
                    xyz_new[i] -= param->enCrossBoundry ? param->sample_length[i] : 2*rnd_wlk;
            }
           
            // ------ subscripts to linear indices ------
            ind = sub2ind(xyz_new[0]*param->scale2grid[0], xyz_new[1]*param->scale2grid[1], xyz_new[2]*param->scale2grid[2], param->fieldmap_size[0], param->fieldmap_size[1]);
            if(ind > param->matrix_length)
            {
                printf("Error:spin=%d, ind=%llu, %d,  scale=(%f), xyz_new=(%f, %f, %f)\n",spin_no, ind, current_timepoint, param->scale2grid[0], xyz_new[0], xyz_new[1], xyz_new[2]);
                return;
            }
            // ------ accumulate phase ------
            if(ind != ind_old) // fewer access to the global memory which is slow. Helpful for large samples!
            {   
                // cross-tissue diffusion
                ts = pMask[ind];           
                if (ts != ts_old) 
                    if (dist_cross_tissue(gen_u) >= param->pXY[ts_old*param->n_tissue_type + ts])
                    {
                        if(itr++ > param->max_iterations)
                        {
                            printf("Warning! spin %d is stuck at (%f, %f, %f) and is considered lost (dummy=%d time=%d).\n", spin_no, xyz_new[0], xyz_new[1], xyz_new[2], dummy_scan, current_timepoint);
                            return;
                        }
                        continue;
                    }
                    else
                        ts_old = ts;
                itr = 0;       
                field = pFieldMap[ind_old = ind];
                ind = pMask[ind]; // the index of the tissue type
                T1 = param->T1[ind];
                T2 = param->T2[ind];
            }     
            accumulated_phase += field;
         
            // ------ apply dephasing if there is any ------
            if(counter_dephasing < param->n_dephasing && param->dephasing_T[counter_dephasing] == current_timepoint)
            {
                accumulated_phase += (float)spin_no * param->dephasing[counter_dephasing] / (float)param->n_spins; // assign dephasing linearly to spins 
                counter_dephasing++;
            }

            // ------ apply gradient if there is any ------
            if(counter_gradient < param->n_gradient && param->gradient_T[counter_gradient] == current_timepoint)
            {
                const float *Gxyz = param->gradient_xyz + 3*counter_gradient;
                accumulated_phase +=  (Gxyz[0]*xyz_new[0] + Gxyz[1]*xyz_new[1] + Gxyz[2]*xyz_new[2]) * param->dt*GAMMA*RAD2DEG; //  Gx * x + Gy * y + Gz * z
                counter_gradient++;
            }
                 
            // ------ apply other RF pulse if there is any ------
            if(current_rf < param->n_RF && param->RF_ST[current_rf] == current_timepoint)
            {
                // dephase and relax    
                time_elapsed = (current_timepoint - old_timepoint) * param->dt;
                dephase_relax(m0, m1, accumulated_phase, T1, T2, time_elapsed);
                // apply RF pulse
                xrot_withphase (param->RF_FA[current_rf], param->RF_PH[current_rf], m1, m0); // Note m0 and m1 are swapped here, so that we can use m0 for the next iteration
                accumulated_phase = 0; // reset phase since we have it now applied
                old_timepoint = current_timepoint;
                current_rf++;
            }

            // ------ echoes are only recorded in the last scan ------
            if (is_lastscan && current_te < param->n_TE && param->TE[current_te] == current_timepoint)
            {
                // dephase and relax                
                time_elapsed = (current_timepoint - old_timepoint) * param->dt;
                dephase_relax(m0, m1, accumulated_phase, T1, T2, time_elapsed);
                // save echo and copy m1 to m0 for the next iteration
                for (uint32_t i=0, shift=3*param->n_TE*spin_no + 3*current_te; i<3; i++)
                    M1[shift + i] = m0[i] = m1[i];
                T[spin_no*param->n_TE + current_te] = ts;

                accumulated_phase = 0; // reset phase since we have applied it in the previous step
                old_timepoint = current_timepoint;
                current_te++;
            }

            // update old position with the new one
            if(param->enRecordTrajectory && (current_timepoint != 0 || dummy_scan != 0))
                xyz1 += 3;      
            for (uint8_t i=0; i < 3; i++)
                xyz1[i] = xyz_new[i];
            // increase timepoint
            current_timepoint++;            
        }
        // dephase and relax    
        time_elapsed = (current_timepoint - old_timepoint) * param->dt;
        dephase_relax(m0, m1, accumulated_phase, T1, T2, time_elapsed);

        // copy m1 to m0 for the next iteration
        for(uint8_t i=0; i<3; i++)
            m0[i] = m1[i];
    }
}


//---------------------------------------------------------------------------------------------
//  
//---------------------------------------------------------------------------------------------

__global__ void cu_scalePos(float *scaled_xyz, float *initial_xyz, float scale, uint64_t size)
{
    uint64_t n = blockIdx.x * blockDim.x + threadIdx.x ;
    if(n < size)
    {
        uint64_t ind = 3*n;
        scaled_xyz[ind+0] = initial_xyz[ind+0] * scale;
        scaled_xyz[ind+1] = initial_xyz[ind+1] * scale;
        scaled_xyz[ind+2] = initial_xyz[ind+2] * scale;
    }
}

//---------------------------------------------------------------------------------------------
// CUDA kernel to perform array multiplication with a constant
//---------------------------------------------------------------------------------------------
__global__ void cu_scaleArray(float *array, float scale, uint64_t size)
{
    auto n = blockIdx.x * blockDim.x + threadIdx.x ;
    if(n < size)
        array[n] *= scale;
}

//---------------------------------------------------------------------------------------------
// CUDA kernel to generate random initial position
//---------------------------------------------------------------------------------------------

__global__ void cu_randPosGen(float *spin_position_xyz, simulation_parameters *param, const uint8_t *pMask, uint32_t spin_no)
{
    spin_no = blockIdx.x * blockDim.x + threadIdx.x ;
    if(spin_no >= param->n_spins)
        return;

    thrust::minstd_rand  gen(param->seed + spin_no);
    thrust::uniform_real_distribution<float> dist_initial_point(0.f, 1.f);
    gen.discard(param->seed + spin_no);

    float *xyz = spin_position_xyz + 3*spin_no;
    for (uint8_t i = 0; i < 3; i++)
        xyz[i] = dist_initial_point(gen) * param->sample_length[i];
}

//---------------------------------------------------------------------------------------------
//  check for CUDA and GPU device
//---------------------------------------------------------------------------------------------
void print_device_info()
{
    const int kb = 1024;
    const int mb = kb * kb;
    size_t free, total;
    
    int32_t cuda_version, driver_version;
    int32_t device_count = getDeviceCount();
    hipRuntimeGetVersion(&cuda_version);
    hipDriverGetVersion(&driver_version);
    std::cout << "\nDriver version: "<< driver_version << ", CUDA version: "<< cuda_version << ", Number of devices: " << device_count << std::endl;

    hipDeviceProp_t device_properties;
    for(int i=0; i<device_count; i++)
    {
        hipSetDevice(i);
        hipGetDeviceProperties(&device_properties, i);
        std::cout << "Device " << i+1 << ", " << device_properties.name << std::endl;
        // hipDeviceReset();
        hipMemGetInfo(&free, &total);
        std::cout << "Free GPU memory: " << free / mb << " MB (out of " << total / mb << " MB)" << std::endl;
    }
}

uint32_t getDeviceCount()
{
    int32_t device_count;
    checkCudaErrors(hipGetDeviceCount(&device_count));
    return device_count;
}

bool check_memory_size(size_t required_size_MB)
{
    size_t free, total;
    bool memory_ok = true;
    int32_t device_count = getDeviceCount();
    hipDeviceProp_t device_properties;
    std::cout << "Device(s) memeory check:" << '\n';
    for(int i=0; i<device_count; i++)
    {
        hipSetDevice(i);
        hipGetDeviceProperties(&device_properties, i);
        hipMemGetInfo(&free, &total);
        std::cout << "  Device " << i+1 << ", " << device_properties.name  << ": " << (free>required_size_MB ? "OK" : "Not enough") << '\n';
        if(free<required_size_MB)
            BOOST_LOG_TRIVIAL(fatal) << "Not enough GPU memory for the simulation in device "<< i <<"! Required=" << required_size_MB <<" MB, Available=" << free << " MB";
        memory_ok = free<required_size_MB ? false:memory_ok;
    }
    return memory_ok;
}
