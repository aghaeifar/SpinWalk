/* --------------------------------------------------------------------------
 * Project: Microvascular
 * File: microvascular_gpu.cu
 *
 * Author   : Ali Aghaeifar <ali.aghaeifar@tuebingen.mpg.de>
 * Date     : 10.02.2023
 * Descrip  : simulating BOLD in microvascular network
 * -------------------------------------------------------------------------- */

// compile :  nvcc microvascular_gpu.cu -Xptxas -v -O3  -arch=compute_86 -code=sm_86  -Xcompiler -fopenmp

#include <random>
#include <filesystem>

#include "./common/kernels.h"
#include "./common/reader.h"


#define CONFIG_DEFAULT     "./inputs/config_default.ini"
#define THREADS_PER_BLOCK  64

using namespace std;

int main(int argc, char * argv[])
{
    std::vector<std::string> config_files(1, CONFIG_DEFAULT);
    if(argc > 2)
    {
        std::cout << "Usage: " << argv[0] << " <config_file>" << std::endl;
        return 1;
    }
    if(argc == 2)
        config_files.push_back(argv[1]);

    map<string, vector<string> > filenames = {{"fieldmap", vector<string>()},
                                              {"output", vector<string>()},
                                              {"xyz0", vector<string>()},
                                              {"m0", vector<string>()} }; 
    std::vector<float> sample_length_scales, fieldmap;
    std::vector<char> mask;
    simulation_parameters param;

    // ========== read config file ==========
    param.fieldmap_size[0] = param.fieldmap_size[1] = param.fieldmap_size[2] = 0;
    param.sample_length[0] = param.sample_length[1] = param.sample_length[2] = 0.f;
    for(uint8_t cnf_fl=0; cnf_fl<config_files.size(); cnf_fl++)
        if(reader::read_config(config_files[cnf_fl], param, sample_length_scales, filenames) == false)
        {
            std::cout << "Reading config file failed. Aborting...!" << std::endl;
            return 1;
        }

    if (param.seed == 0)
        param.seed = std::random_device{}();

    param.n_timepoints = param.TR / param.dt; // includes start point

    // ========== simulating steady-state signal ==========
    if(param.enSteadyStateSimulation && param.n_dummy_scan != 0)
    {
        simulate_steady_state(param);
        std::cout<< std::string(30, '-')  << std::endl;
    }

    // ========== Dump Settings ==========
    if(param.enDebug)
    {
        std::cout << "Dumping settings:" << std::endl;
        for (int32_t i = 0; i < param.n_fieldmaps; i++)
            std::cout << "Fieldmap " << i << " = " << filenames.at("fieldmap")[i] << std::endl;
        
        for (int32_t i = 0; i < param.n_sample_length_scales; i++)
            std::cout << "Sample length scale " << i << " = " << sample_length_scales[i] << std::endl;

        param.dump();
        std::cout<< std::string(30, '-')  << std::endl;
    }

    // ========== checking GPU(s) ==========
    int32_t device_count;
    checkCudaErrors(hipGetDeviceCount(&device_count));
    std::cout << "Number of GPU(s): " << device_count << std::endl;
    param.n_spins /= device_count; // spins will be distributed in multiple GPUs (if there is). We hope it is divisible 
    int32_t numBlocks = (param.n_spins + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    std::vector<float> M0(3 * param.n_spins * param.n_sample_length_scales * device_count, 0.f);
    std::vector<float> M1(3 * param.n_spins * param.n_sample_length_scales * device_count, 0.f);
    std::vector<float> XYZ0(3 * param.n_spins * param.n_sample_length_scales * device_count, 0.f);
    std::vector<float> XYZ1(3 * param.n_spins * param.n_sample_length_scales * device_count, 0.f);

    for (int16_t fieldmap_no=0; fieldmap_no<param.n_fieldmaps; fieldmap_no++)
    {
        // ========== load field-maps ==========
        std::string fieldmap_file = filenames.at("fieldmap")[fieldmap_no];
        reader::read_fieldmap(fieldmap_file, fieldmap, mask, param);

        for(int i=0; i<3; i++)
            param.scale2grid[i] = (param.fieldmap_size[i] - 1.) / param.sample_length[i];
         
        // ========== distributing between devices ==========
        std::vector<float *> d_pFieldMap(device_count, NULL), d_position_start(device_count, NULL), d_position_start_scaled(device_count, NULL);
        std::vector<float *> d_M1(device_count, NULL), d_XYZ1(device_count, NULL);
        std::vector<bool *> d_pMask(device_count, NULL);
        std::vector<simulation_parameters *> d_param(device_count, NULL);
        std::vector<hipStream_t> streams(device_count, NULL);

        #pragma omp parallel for
        for(int32_t d=0; d<device_count; d++)
        {
            checkCudaErrors(hipSetDevice(d));            
            checkCudaErrors(hipStreamCreate(&streams[d]));

            checkCudaErrors(hipMalloc((void**)&d_param[d],             sizeof(simulation_parameters)));
            checkCudaErrors(hipMalloc((void**)&d_pFieldMap[d],         sizeof(fieldmap[0]) * fieldmap.size()));   
            checkCudaErrors(hipMalloc((void**)&d_pMask[d],             sizeof(mask[0]) * mask.size())); 
            checkCudaErrors(hipMalloc((void**)&d_position_start[d],    sizeof(float) * param.n_spins * 3));
            checkCudaErrors(hipMalloc((void**)&d_position_start_scaled[d],sizeof(float) * param.n_spins * 3));
            checkCudaErrors(hipMalloc((void**)&d_M1[d],                sizeof(float) * param.n_spins * 3));
            checkCudaErrors(hipMalloc((void**)&d_XYZ1[d],              sizeof(float) * param.n_spins * 3));
            
            checkCudaErrors(hipMemcpyAsync(d_pFieldMap[d], fieldmap.data(), fieldmap.size() * sizeof(fieldmap[0]), hipMemcpyHostToDevice, streams[d]));
            checkCudaErrors(hipMemcpyAsync(d_pMask[d],     mask.data(),     mask.size() * sizeof(mask[0]),         hipMemcpyHostToDevice, streams[d]));
            checkCudaErrors(hipMemcpyAsync(d_param[d],     &param,          sizeof(simulation_parameters),         hipMemcpyHostToDevice, streams[d]));

            // generate initial spatial position for spins, based on sample_length_ref
            printf("GPU %d) Generating random initial position for spins... ", d);
            generate_initial_position<<<numBlocks, THREADS_PER_BLOCK, 0, streams[d]>>>(d_position_start[d], d_param[d], d_pMask[d]);
            gpuCheckKernelExecutionError( __FILE__, __LINE__);
            printf("Done!\n");
        }

        // ========== run ==========        
        hipEvent_t start;
        hipEvent_t end;
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&end));
        checkCudaErrors(hipEventRecord(start));
        
        simulation_parameters param_local;
        memcpy(&param_local, &param, sizeof(simulation_parameters));
        for (int32_t sl = 0; sl < param.n_sample_length_scales; sl++)
        {
            for (int i = 0; i < 3; i++)
            {
                param_local.sample_length[i] = sample_length_scales[sl] * param.sample_length[i];
                param_local.scale2grid[i] = (param_local.fieldmap_size[i] - 1.) / param_local.sample_length[i];
            }

            #pragma omp parallel for
            for (int32_t d = 0; d < device_count; d++)
            {
                if (param.n_sample_length_scales > 1)
                    printf("GPU %d) Simulating sample scale %2d = %8.5f\n", d, sl, sample_length_scales[sl]);
                checkCudaErrors(hipSetDevice(d));
                hipMemcpy(d_param[d], &param_local, sizeof(simulation_parameters), hipMemcpyHostToDevice);

                scale_initial_positions << <numBlocks, THREADS_PER_BLOCK, 0, streams[d] >> > (d_position_start_scaled[d], d_position_start[d], sample_length_scales[sl], param.n_spins);
                gpuCheckKernelExecutionError(__FILE__, __LINE__);

                simulation_kernel << <numBlocks, THREADS_PER_BLOCK, 0, streams[d] >> > (d_param[d], d_pFieldMap[d], d_pMask[d], d_position_start_scaled[d], d_M1[d], d_XYZ1[d]);
                gpuCheckKernelExecutionError(__FILE__, __LINE__);

                int shift = 3*param.n_spins*device_count*sl + 3*param.n_spins*d;
                checkCudaErrors(hipMemcpyAsync(M1.data()   + shift, d_M1[d]  , sizeof(float) * 3 * param.n_spins, hipMemcpyDeviceToHost, streams[d]));
                checkCudaErrors(hipMemcpyAsync(XYZ1.data() + shift, d_XYZ1[d], sizeof(float) * 3 * param.n_spins, hipMemcpyDeviceToHost, streams[d]));
            }
        }

        float elapsedTime;
        checkCudaErrors(hipEventRecord(end));
        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, end));
        std::cout << "Entire simulation over " << device_count << " GPU(s) took " << std::fixed << std::setprecision(2) << elapsedTime/1000. << " second(s)" << std::endl;

        // ========== clean up GPU ==========
        #pragma omp parallel for
        for(int32_t d=0; d<device_count; d++)
        {
            checkCudaErrors(hipSetDevice(d));   
            checkCudaErrors(hipFree(d_param[d]));
            checkCudaErrors(hipFree(d_pFieldMap[d]));
            checkCudaErrors(hipFree(d_pMask[d]));
            checkCudaErrors(hipFree(d_position_start[d]));
            checkCudaErrors(hipFree(d_position_start_scaled[d]));
            checkCudaErrors(hipFree(d_M1[d]));
            checkCudaErrors(hipFree(d_XYZ1[d]));
            checkCudaErrors(hipStreamDestroy(streams[d]));
            checkCudaErrors(hipEventDestroy(start));
            checkCudaErrors(hipEventDestroy(end));
        }
        
        // ========== save results ========== 
        std::string append = std::to_string(fieldmap_no) + "_" + std::filesystem::path(fieldmap_file).stem().string(); // Thanks to C++17, we can use std::filesystem
        output_header hdr(3, param.n_spins, device_count, param.n_sample_length_scales);
        save_output(M1  , filenames.at("output")[0], "M1_"   + append, hdr, sample_length_scales);
        save_output(XYZ1, filenames.at("output")[0], "XYZ1_" + append, hdr, sample_length_scales);

        std::cout << std::string(50, '=') << std::endl;
    }
}

